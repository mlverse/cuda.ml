#include "hip/hip_runtime.h"
#include "async_utils.cuh"
#include "cuda_utils.h"
#include "fil_utils.h"
#include "handle_utils.h"
#include "matrix_utils.h"
#include "pinned_host_vector.h"
#include "preprocessor.h"
#include "stream_allocator.h"
#include "treelite_utils.cuh"

#include <cuml/fil/fil.h>
#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <treelite/c_api.h>

#include <Rcpp.h>

#include <memory>
#include <string>

namespace cuml4r {
namespace {

enum class ModelType { XGBoost, XGBoostJSON, LightGBM };

struct FILModel {
  __host__ FILModel(std::unique_ptr<raft::handle_t> handle,
                    fil::forest_uptr forest, size_t const num_classes)
    : handle_(std::move(handle)),
      forest_(std::move(forest)),
      numClasses_(num_classes) {}

  std::unique_ptr<raft::handle_t> const handle_;
  // NOTE: the destruction of `forest_` must precede the destruction of
  // `handle_`.
  fil::forest_uptr forest_;
  size_t const numClasses_;
};

__host__ int treeliteLoadModel(ModelType const model_type, char const* filename,
                               TreeliteHandle& tl_handle) {
  switch (model_type) {
    case ModelType::XGBoost:
      return TreeliteLoadXGBoostModel(filename, tl_handle.get());
    case ModelType::XGBoostJSON:
      return TreeliteLoadXGBoostJSON(filename, tl_handle.get());
    case ModelType::LightGBM:
      return TreeliteLoadLightGBMModel(filename, tl_handle.get());
  }

  // unreachable
  return -1;
}

/*
 * The 'ML::fil::treelite_params_t::threads_per_tree' and
 * 'ML::fil::treelite_params_t::n_items' parameters are only supported in
 * RAPIDS cuML 21.08 or above.
 */
CUML4R_ASSIGN_IF_PRESENT(threads_per_tree)
CUML4R_NOOP_IF_ABSENT(threads_per_tree)

CUML4R_ASSIGN_IF_PRESENT(n_items)
CUML4R_NOOP_IF_ABSENT(n_items)

}  // namespace

__host__ SEXP fil_load_model(int const model_type, std::string const& filename,
                             int const algo, bool const classification,
                             float const threshold, int const storage_type,
                             int const blocks_per_sm,
                             int const threads_per_tree, int const n_items) {
  Rcpp::List model;

  TreeliteHandle tl_handle;
  {
    auto const rc = treeliteLoadModel(
      /*model_type=*/static_cast<ModelType>(model_type),
      /*filename=*/filename.c_str(), tl_handle);
    if (rc < 0) {
      char const* err = TreeliteGetLastError();
      Rcpp::stop("Failed to load XGBoost model file '%s': %s.",
                 filename.c_str(), err);
    }
  }

  ML::fil::treelite_params_t params;
  params.algo = static_cast<ML::fil::algo_t>(algo);
  params.output_class = classification;
  params.threshold = threshold;
  params.storage_type = static_cast<ML::fil::storage_type_t>(storage_type);
  params.blocks_per_sm = blocks_per_sm;
  params.output_class = classification;
  set_threads_per_tree(params, threads_per_tree);
  set_n_items(params, n_items);
  params.pforest_shape_str = nullptr;

  auto stream_view = stream_allocator::getOrCreateStream();
  auto handle = std::make_unique<raft::handle_t>();
  handle_utils::initializeHandle(*handle, stream_view.value());

  auto forest = fil::make_forest(*handle, /*src=*/[&] {
    ML::fil::forest* f;
    ML::fil::from_treelite(/*handle=*/*handle, /*pforest=*/&f,
                           /*model=*/*tl_handle.get(),
                           /*tl_params=*/&params);
    return f;
  });

  size_t num_classes = 0;
  if (classification) {
    auto const rc = TreeliteQueryNumClass(/*handle=*/*tl_handle.get(),
                                          /*out=*/&num_classes);
    if (rc < 0) {
      char const* err = TreeliteGetLastError();
      Rcpp::stop("TreeliteQueryNumClass failed: %s.", err);
    }

    // Treelite returns 1 as number of classes for binary classification.
    num_classes = std::max(num_classes, size_t(2));
  }

  return Rcpp::XPtr<FILModel>(
    std::make_unique<FILModel>(
      /*handle=*/std::move(handle), std::move(forest), num_classes)
      .release());
}

__host__ int fil_get_num_classes(SEXP const& model) {
  auto const model_xptr = Rcpp::XPtr<FILModel>(model);
  return model_xptr->numClasses_;
}

__host__ Rcpp::NumericMatrix fil_predict(
  SEXP const& model, Rcpp::NumericMatrix const& x,
  bool const output_class_probabilities) {
  auto const model_xptr = Rcpp::XPtr<FILModel>(model);
  auto const m = Matrix<float>(x, /*transpose=*/false);

  if (output_class_probabilities && model_xptr->numClasses_ == 0) {
    Rcpp::stop(
      "'output_class_probabilities' is not applicable for regressions!");
  }

  auto& handle = *(model_xptr->handle_);

  // ensemble input data
  auto const& h_x = m.values;
  thrust::device_vector<float> d_x(h_x.size());
  auto CUML4R_ANONYMOUS_VARIABLE(x_h2d) =
    async_copy(handle.get_stream(), h_x.cbegin(), h_x.cend(), d_x.begin());

  // ensemble output
  thrust::device_vector<float> d_preds(output_class_probabilities
                                         ? model_xptr->numClasses_ * m.numRows
                                         : m.numRows);

  ML::fil::predict(/*h=*/handle, /*f=*/model_xptr->forest_.get(),
                   /*preds=*/d_preds.data().get(),
                   /*data=*/d_x.data().get(), /*num_rows=*/m.numRows,
                   /*predict_proba=*/output_class_probabilities);

  pinned_host_vector<float> h_preds(d_preds.size());
  auto CUML4R_ANONYMOUS_VARIABLE(preds_d2h) = async_copy(
    handle.get_stream(), d_preds.cbegin(), d_preds.cend(), h_preds.begin());

  CUDA_RT_CALL(hipStreamSynchronize(handle.get_stream()));

  return Rcpp::transpose(Rcpp::NumericMatrix(
    output_class_probabilities ? model_xptr->numClasses_ : 1, m.numRows,
    h_preds.begin()));
}

}  // namespace cuml4r
