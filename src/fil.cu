#include "hip/hip_runtime.h"
#include "async_utils.cuh"
#include "cuda_utils.h"
#include "handle_utils.h"
#include "matrix_utils.h"
#include "pinned_host_vector.h"
#include "preprocessor.h"
#include "stream_allocator.h"

#include <cuml/fil/fil.h>
#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <treelite/c_api.h>

#include <Rcpp.h>

#include <memory>
#include <string>

namespace {

enum class ModelType { XGBoost, XGBoostJSON, LightGBM };

__host__ int treeliteLoadModel(ModelType const model_type, char const* filename,
                               ModelHandle& model_handle) {
  switch (model_type) {
    case ModelType::XGBoost:
      return TreeliteLoadXGBoostModel(filename, &model_handle);
    case ModelType::XGBoostJSON:
      return TreeliteLoadXGBoostJSON(filename, &model_handle);
    case ModelType::LightGBM:
      return TreeliteLoadLightGBMModel(filename, &model_handle);
  }

  // unreachable
  return -1;
}

struct TreeliteModel {
  __host__ TreeliteModel(std::unique_ptr<raft::handle_t> handle,
                         ML::fil::forest_t const forest,
                         ModelHandle const model, size_t const num_classes)
    : handle_(std::move(handle)),
      forest_(forest),
      model_(model),
      numClasses_(num_classes) {}
  ~TreeliteModel() {
    if (forest_ != nullptr) {
      ML::fil::free(*handle_, forest_);
    }
  }

  std::unique_ptr<raft::handle_t> const handle_;
  ML::fil::forest_t const forest_;
  ModelHandle const model_;
  size_t const numClasses_;
};

}  // namespace

namespace cuml4r {

__host__ SEXP fil_load_model(int const model_type, std::string const& filename,
                             int const algo, bool const classification,
                             float const threshold, int const storage_type,
                             int const blocks_per_sm,
                             int const threads_per_tree, int const n_items) {
  Rcpp::List model;

  ModelHandle model_handle;
  {
    auto const rc = treeliteLoadModel(
      /*model_type=*/static_cast<ModelType>(model_type),
      /*filename=*/filename.c_str(), model_handle);
    if (rc < 0) {
      char const* err = TreeliteGetLastError();
      Rcpp::stop("Failed to load XGBoost model file '%s': %s.",
                 filename.c_str(), err);
    }
  }

  ML::fil::treelite_params_t params;
  params.algo = static_cast<ML::fil::algo_t>(algo);
  params.output_class = classification;
  params.threshold = threshold;
  params.storage_type = static_cast<ML::fil::storage_type_t>(storage_type);
  params.blocks_per_sm = blocks_per_sm;
  params.threads_per_tree = threads_per_tree;
  params.n_items = n_items;
  params.pforest_shape_str = nullptr;

  auto stream_view = cuml4r::stream_allocator::getOrCreateStream();
  auto handle = std::make_unique<raft::handle_t>();
  cuml4r::handle_utils::initializeHandle(*handle, stream_view.value());

  ML::fil::forest_t forest;

  ML::fil::from_treelite(/*handle=*/*handle, /*pforest=*/&forest,
                         /*model=*/model_handle, /*tl_params=*/&params);

  size_t num_classes = 0;
  if (classification) {
    auto const rc =
      TreeliteQueryNumClass(/*handle=*/model_handle, /*out=*/&num_classes);
    if (rc < 0) {
      char const* err = TreeliteGetLastError();
      Rcpp::stop("TreeliteQueryNumClass failed: %s.", err);
    }

    // Treelite returns 1 as number of classes for binary classification.
    num_classes = std::max(num_classes, size_t(2));
  }

  return Rcpp::XPtr<TreeliteModel>(std::make_unique<TreeliteModel>(
                                     /*handle=*/std::move(handle), forest,
                                     /*model=*/model_handle, num_classes)
                                     .release());
}

__host__ int fil_get_num_classes(SEXP const& model) {
  auto const model_xptr = Rcpp::XPtr<TreeliteModel>(model);
  return model_xptr->numClasses_;
}

__host__ Rcpp::NumericMatrix fil_predict(
  SEXP const& model, Rcpp::NumericMatrix const& x,
  bool const output_class_probabilities) {
  auto const model_xptr = Rcpp::XPtr<TreeliteModel>(model);
  auto const m = cuml4r::Matrix<float>(x, /*transpose=*/false);

  if (output_class_probabilities && model_xptr->numClasses_ == 0) {
    Rcpp::stop(
      "'output_class_probabilities' is not applicable for regressions!");
  }

  auto& handle = *(model_xptr->handle_);

  // ensemble input data
  auto const& h_x = m.values;
  thrust::device_vector<float> d_x(h_x.size());
  auto CUML4R_ANONYMOUS_VARIABLE(x_h2d) = cuml4r::async_copy(
    handle.get_stream(), h_x.cbegin(), h_x.cend(), d_x.begin());

  // ensemble output
  thrust::device_vector<float> d_preds(output_class_probabilities
                                         ? model_xptr->numClasses_ * m.numRows
                                         : m.numRows);

  ML::fil::predict(/*h=*/handle, /*f=*/model_xptr->forest_,
                   /*preds=*/d_preds.data().get(),
                   /*data=*/d_x.data().get(), /*num_rows=*/m.numRows,
                   /*predict_proba=*/output_class_probabilities);

  cuml4r::pinned_host_vector<float> h_preds(d_preds.size());
  auto CUML4R_ANONYMOUS_VARIABLE(preds_d2h) = cuml4r::async_copy(
    handle.get_stream(), d_preds.cbegin(), d_preds.cend(), h_preds.begin());

  CUDA_RT_CALL(hipStreamSynchronize(handle.get_stream()));

  return Rcpp::transpose(Rcpp::NumericMatrix(
    output_class_probabilities ? model_xptr->numClasses_ : 1, m.numRows,
    h_preds.begin()));
}

}  // namespace cuml4r
