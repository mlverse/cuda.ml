#include "async_utils.cuh"
#include "cuda_utils.h"
#include "handle_utils.h"
#include "knn_detail.h"
#include "matrix_utils.h"
#include "pinned_host_vector.h"
#include "preprocessor.h"
#include "random_forest.cuh"
#include "stream_allocator.h"

#include <raft/spatial/knn/ann_common.h>
#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <cuml/neighbors/knn.hpp>

#include <Rcpp.h>

#include <initializer_list>
#include <memory>
#include <string>
#include <unordered_map>
#include <vector>

namespace cuml4r {
namespace knn {
namespace {

// string constants related to KNN params
char const* const N_LIST = "nlist";
char const* const N_PROBE = "nprobe";
char const* const M_VALUE = "M";
char const* const N_BITS = "n_bits";
char const* const USE_PRE_COMPUTED_TABLES = "usePrecomputedTables";
char const* const Q_TYPE = "qtype";
char const* const ENCODE_RESIDUAL = "encodeResidual";
// string constants related to KNN model attributes
char const* const KNN_INDEX = "knn_index";
char const* const ALGO = "algo";
char const* const P_VALUE = "p";
char const* const METRIC = "metric";
char const* const N_SAMPLES = "n_samples";
char const* const N_DIMS = "n_dims";

std::unordered_map<std::string, raft::spatial::knn::QuantizerType> const
  kQuantizerTypes{
    {"QT_8bit", raft::spatial::knn::QuantizerType::QT_8bit},
    {"QT_4bit", raft::spatial::knn::QuantizerType::QT_4bit},
    {"QT_8bit_uniform", raft::spatial::knn::QuantizerType::QT_8bit_uniform},
    {"QT_4bit_uniform", raft::spatial::knn::QuantizerType::QT_4bit_uniform},
    {"QT_fp16", raft::spatial::knn::QuantizerType::QT_fp16},
    {"QT_8bit_direct", raft::spatial::knn::QuantizerType::QT_8bit_direct},
    {"QT_6bit", raft::spatial::knn::QuantizerType::QT_6bit}};

// Additional info for setting KNN params
struct ParamsDetails {
  int numRows_;
  int numCols_;
};

enum class Algo { BRUTE_FORCE = 0, IVFFLAT = 1, IVFPQ = 2, IVFSQ = 3 };

struct NearestNeighbors {
  NearestNeighbors() {}
  NearestNeighbors(int const n_samples, int const n_neighbors) {
    auto const n_entries = n_samples * n_neighbors;
    indices.resize(n_entries);
    dists.resize(n_entries);
  }

  thrust::device_vector<int64_t> indices;
  thrust::device_vector<float> dists;
};

template <typename ResponseT>
class PredictionCtx {
 public:
  using ResponseVecT = typename knn::detail::RcppVector<ResponseT>::type;
  __host__ PredictionCtx(Rcpp::List const& model, Rcpp::NumericMatrix const& x,
                         int const n_neighbors)
    : nSamples_(x.nrow()),
      nFeatures_(x.ncol()),
      modelKnnIndex_(Rcpp::XPtr<raft::spatial::knn::knnIndex>(
        static_cast<SEXP>(model[KNN_INDEX]))),
      modelAlgoType_(static_cast<knn::Algo>(Rcpp::as<int>(model[ALGO]))),
      modelDistType_(static_cast<raft::distance::DistanceType>(
        Rcpp::as<int>(model[METRIC]))),
      modelP_(Rcpp::as<float>(model[P_VALUE])),
      modelNSamples_(Rcpp::as<int>(model[N_SAMPLES])),
      modelNDims_(Rcpp::as<int>(model[N_DIMS])),
      streamView_(cuml4r::stream_allocator::getOrCreateStream()) {
    cuml4r::handle_utils::initializeHandle(handle_, streamView_.value());
    auto const input_m = cuml4r::Matrix<float>(x, /*transpose=*/false);
    // KNN classifier input
    auto const& h_x = input_m.values;
    dX_.resize(h_x.size());
    xH2D_ = cuml4r::async_copy(streamView_.value(), h_x.cbegin(), h_x.cend(),
                               dX_.begin());

    ResponseVecT const model_resps(
      Rcpp::as<ResponseVecT>(model[detail::kResponses]));
    auto h_y = Rcpp::as<cuml4r::pinned_host_vector<ResponseT>>(model_resps);
    dY_.resize(h_y.size());
    yH2D_ = cuml4r::async_copy(streamView_.value(), h_y.cbegin(), h_y.cend(),
                               dY_.begin());

    nearestNeighbors_ = query_nearest_neighbors(n_neighbors);

    CUDA_RT_CALL(hipStreamSynchronize(streamView_.value()));
  }

  __host__ NearestNeighbors query_nearest_neighbors(int const n_neighbors) {
    NearestNeighbors res(nSamples_, n_neighbors);
    auto d_input = dX_.data().get();

    if (modelAlgoType_ == Algo::BRUTE_FORCE) {
      std::vector<float*> input{d_input};
      std::vector<int> sizes{nSamples_};

      ML::brute_force_knn(handle_, input, sizes, /*D=*/modelNDims_,
                          /*search_items=*/d_input,
                          /*n=*/nSamples_, /*res_I=*/res.indices.data().get(),
                          /*res_D=*/res.dists.data().get(), /*k=*/n_neighbors,
                          /*rowMajorIndex=*/true, /*rowMajorQuery=*/true,
                          /*metric=*/modelDistType_, modelP_);
    } else {
      ML::approx_knn_search(handle_, /*distances=*/res.dists.data().get(),
                            /*indices=*/res.indices.data().get(),
                            /*index=*/modelKnnIndex_.get(), /*k=*/n_neighbors,
                            /*query_array=*/d_input, /*n=*/nSamples_);
    }

    return res;
  }

  // input dimensions
  int const nSamples_;
  int const nFeatures_;
  // attributes from the KNN model object
  Rcpp::XPtr<raft::spatial::knn::knnIndex> const modelKnnIndex_;
  Algo const modelAlgoType_;
  raft::distance::DistanceType const modelDistType_;
  float const modelP_;
  int const modelNSamples_;
  int const modelNDims_;
  // CUDA stream, etc
  rmm::cuda_stream_view streamView_;
  raft::handle_t handle_;
  // KNN classifier inputs
  thrust::device_vector<float> dX_;
  thrust::device_vector<ResponseT> dY_;
  NearestNeighbors nearestNeighbors_;

 private:
  cuml4r::unique_marker xH2D_;
  cuml4r::unique_marker yH2D_;
};

__host__ void validate_param_list(
  Rcpp::List const& params, std::initializer_list<char const*> required_attrs) {
  for (auto const attr : required_attrs) {
    if (!params.containsElementNamed(attr)) {
      Rcpp::stop("Required attribute '%s' is missing from algo params!", attr);
    }
  }
}

__host__ void validate_algo_params(Algo const algo, Rcpp::List const& params) {
  if (algo == Algo::IVFFLAT) {
    validate_param_list(params, {N_LIST, N_PROBE});
  } else if (algo == Algo::IVFPQ) {
    validate_param_list(
      params, {N_LIST, N_PROBE, M_VALUE, N_BITS, USE_PRE_COMPUTED_TABLES});
  } else if (algo == Algo::IVFSQ) {
    validate_param_list(params, {N_LIST, N_PROBE, Q_TYPE, ENCODE_RESIDUAL});
  }
}

__host__ std::unique_ptr<raft::spatial::knn::knnIndexParam>
build_ivfflat_algo_params(Rcpp::List params, bool const automated) {
  if (automated) {
    params[N_LIST] = 8;
    params[N_PROBE] = 2;
  }

  auto algo_params = std::make_unique<raft::spatial::knn::IVFFlatParam>();
  algo_params->nlist = params[N_LIST];
  algo_params->nprobe = params[N_PROBE];

  return algo_params;
}

__host__ std::unique_ptr<raft::spatial::knn::knnIndexParam>
build_ivfpq_algo_params(Rcpp::List params, bool const automated,
                        ParamsDetails const& details) {
  constexpr std::array<int, 13> kAllowedSubquantizers = {
    1, 2, 3, 4, 8, 12, 16, 20, 24, 28, 32, 40, 48};
  constexpr std::array<int, 13> kAllowedSubDimSize = {1,  2,  3,  4,  6,  8, 10,
                                                      12, 16, 20, 24, 28, 32};

  if (automated) {
    auto const n = details.numRows_;
    auto const d = details.numCols_;

    params[N_LIST] = 8;
    params[N_PROBE] = 3;

    for (auto const n_subq : kAllowedSubquantizers) {
      if (d % n_subq == 0 &&
          std::find(kAllowedSubDimSize.cbegin(), kAllowedSubDimSize.cend(),
                    d / n_subq) != kAllowedSubDimSize.cend()) {
        params[USE_PRE_COMPUTED_TABLES] = false;
        params[M_VALUE] = n_subq;
        break;
      }
    }

    if (!params.containsElementNamed(M_VALUE)) {
      for (auto const n_subq : kAllowedSubquantizers) {
        if (d % n_subq == 0) {
          params[USE_PRE_COMPUTED_TABLES] = true;
          params[M_VALUE] = n_subq;
          break;
        }
      }
    }

    params[N_BITS] = 4;
    for (auto const n_bits : {8, 6, 5}) {
      auto const min_train_points = (1 << n_bits) * 39;
      if (n >= min_train_points) {
        params[N_BITS] = n_bits;
        break;
      }
    }
  }

  auto algo_params = std::make_unique<raft::spatial::knn::IVFPQParam>();
  algo_params->nlist = Rcpp::as<int>(params[N_LIST]);
  algo_params->nprobe = Rcpp::as<int>(params[N_PROBE]);
  algo_params->M = Rcpp::as<int>(params[M_VALUE]);
  algo_params->n_bits = Rcpp::as<int>(params[N_BITS]);
  algo_params->usePrecomputedTables =
    Rcpp::as<bool>(params[USE_PRE_COMPUTED_TABLES]);

  return algo_params;
}

__host__ std::unique_ptr<raft::spatial::knn::knnIndexParam>
build_ivfsq_algo_params(Rcpp::List params, bool const automated) {
  if (automated) {
    params[N_LIST] = 8;
    params[N_PROBE] = 2;
    params[Q_TYPE] = "QT_8bit";
    params[ENCODE_RESIDUAL] = true;
  }

  auto algo_params = std::make_unique<raft::spatial::knn::IVFSQParam>();
  algo_params->nlist = Rcpp::as<int>(params[N_LIST]);
  algo_params->nprobe = Rcpp::as<int>(params[N_PROBE]);
  auto const qtype = Rcpp::as<std::string>(params[Q_TYPE]);
  {
    auto const qtype_iter = kQuantizerTypes.find(qtype);
    if (kQuantizerTypes.cend() == qtype_iter) {
      Rcpp::stop("Unsupported quantizer type '" + qtype + "'");
    }
    algo_params->qtype = qtype_iter->second;
  }
  algo_params->encodeResidual = Rcpp::as<bool>(params[ENCODE_RESIDUAL]);

  return algo_params;
}

__host__ std::unique_ptr<raft::spatial::knn::knnIndexParam> build_algo_params(
  Algo const algo, Rcpp::List const& params, ParamsDetails const& details) {
  bool const automated = (params.size() == 0);

  if (!automated) {
    validate_algo_params(algo, params);
  }

  switch (algo) {
    case Algo::IVFFLAT:
      return build_ivfflat_algo_params(params, automated);
    case Algo::IVFPQ:
      return build_ivfpq_algo_params(params, automated, details);
    case Algo::IVFSQ:
      return build_ivfsq_algo_params(params, automated);
    default:
      return nullptr;
  }
}

__host__ std::unique_ptr<raft::spatial::knn::knnIndex> build_knn_index(
  raft::handle_t& handle, float* const d_input, int const n_samples,
  int const n_features, Algo const algo_type,
  raft::distance::DistanceType const dist_type, float const p,
  Rcpp::List const& algo_params) {
  std::unique_ptr<raft::spatial::knn::knnIndex> knn_index(nullptr);

  if (algo_type == Algo::IVFFLAT || algo_type == Algo::IVFPQ ||
      algo_type == Algo::IVFSQ) {
    ParamsDetails details;
    details.numRows_ = n_samples;
    details.numCols_ = n_features;

    auto params =
      build_algo_params(/*algo=*/algo_type, /*params=*/algo_params, details);

    knn_index = std::make_unique<raft::spatial::knn::knnIndex>();
    ML::approx_knn_build_index(handle,
                               /*index=*/knn_index.get(),
                               /*params=*/params.get(),
                               /*metric=*/dist_type,
                               /*metricArg=*/p,
                               /*index_array=*/d_input,
                               /*n=*/n_samples,
                               /*D=*/n_features);

    CUDA_RT_CALL(hipStreamSynchronize(handle.get_stream()));
  }

  return knn_index;
}

}  // namespace
}  // namespace knn

__host__ Rcpp::List knn_fit(Rcpp::NumericMatrix const& x, int const algo,
                            int const metric, float const p,
                            Rcpp::List const& algo_params) {
  auto const algo_type = static_cast<knn::Algo>(algo);
  auto const dist_type = static_cast<raft::distance::DistanceType>(metric);

  auto const input_m = cuml4r::Matrix<float>(x, /*transpose=*/false);
  int const n_samples = input_m.numRows;
  int const n_features = input_m.numCols;

  auto stream_view = cuml4r::stream_allocator::getOrCreateStream();
  raft::handle_t handle;
  cuml4r::handle_utils::initializeHandle(handle, stream_view.value());

  // knn input
  auto const& h_x = input_m.values;
  thrust::device_vector<float> d_x(h_x.size());
  auto CUML4R_ANONYMOUS_VARIABLE(x_h2d) = cuml4r::async_copy(
    stream_view.value(), h_x.cbegin(), h_x.cend(), d_x.begin());

  auto knn_index =
    build_knn_index(handle, /*d_input=*/d_x.data().get(), n_samples, n_features,
                    algo_type, dist_type, p, algo_params);

  Rcpp::List model;
  model[knn::KNN_INDEX] =
    Rcpp::XPtr<raft::spatial::knn::knnIndex>(knn_index.release());
  model[knn::ALGO] = algo;
  model[knn::METRIC] = metric;
  model[knn::P_VALUE] = p;
  model[knn::N_SAMPLES] = n_samples;
  model[knn::N_DIMS] = n_features;

  return model;
}

__host__ Rcpp::IntegerVector knn_classifier_predict(
  Rcpp::List const& model, Rcpp::NumericMatrix const& x,
  int const n_neighbors) {
  // KNN classifier input & pre-processing
  knn::PredictionCtx<int> ctx(model, x, n_neighbors);
  std::vector<int*> y_vec{ctx.dY_.data().get()};

  // KNN classifier output
  thrust::device_vector<int> d_out(ctx.nSamples_);

  ML::knn_classify(/*handle=*/ctx.handle_, /*out=*/d_out.data().get(),
                   /*knn_indices=*/ctx.nearestNeighbors_.indices.data().get(),
                   /*y=*/y_vec, /*n_index_rows=*/ctx.modelNSamples_,
                   /*n_query_rows=*/ctx.nSamples_, /*k=*/n_neighbors);

  cuml4r::pinned_host_vector<int> h_out(d_out.size());
  auto CUML4R_ANONYMOUS_VARIABLE(out_d2h) = cuml4r::async_copy(
    ctx.streamView_.value(), d_out.cbegin(), d_out.cend(), h_out.begin());
  CUDA_RT_CALL(hipStreamSynchronize(ctx.streamView_.value()));

  return Rcpp::IntegerVector(h_out.cbegin(), h_out.cend());
}

__host__ Rcpp::NumericMatrix knn_classifier_predict_probabilities(
  Rcpp::List const& model, Rcpp::NumericMatrix const& x,
  int const n_neighbors) {
  // KNN classifier input & pre-processing
  knn::PredictionCtx<int> ctx(model, x, n_neighbors);
  std::vector<int*> y_vec{ctx.dY_.data().get()};
  int const n_classes =
    Rcpp::unique(Rcpp::as<Rcpp::IntegerVector>(model[knn::detail::kResponses]))
      .size();

  // KNN classifier output
  thrust::device_vector<float> d_out(ctx.nSamples_ * n_classes);
  std::vector<float*> out_vec{d_out.data().get()};

  ML::knn_class_proba(
    /*handle=*/ctx.handle_, /*out=*/out_vec,
    /*knn_indices=*/ctx.nearestNeighbors_.indices.data().get(),
    /*y=*/y_vec, /*n_index_rows=*/ctx.modelNSamples_,
    /*n_query_rows=*/ctx.nSamples_, /*k=*/n_neighbors);

  cuml4r::pinned_host_vector<float> h_out(d_out.size());
  auto CUML4R_ANONYMOUS_VARIABLE(out_d2h) = cuml4r::async_copy(
    ctx.streamView_.value(), d_out.cbegin(), d_out.cend(), h_out.begin());
  CUDA_RT_CALL(hipStreamSynchronize(ctx.streamView_.value()));

  return Rcpp::transpose(
    Rcpp::NumericMatrix(n_classes, ctx.nSamples_, h_out.begin()));
}

Rcpp::NumericVector knn_regressor_predict(Rcpp::List const& model,
                                          Rcpp::NumericMatrix const& x,
                                          int const n_neighbors) {
  // KNN regressor input & pre-processing
  knn::PredictionCtx<float> ctx(model, x, n_neighbors);
  std::vector<float*> y_vec{ctx.dY_.data().get()};

  // KNN regressor output
  thrust::device_vector<float> d_out(ctx.nSamples_);

  ML::knn_regress(/*handle=*/ctx.handle_, /*out=*/d_out.data().get(),
                  /*knn_indices=*/ctx.nearestNeighbors_.indices.data().get(),
                  /*y=*/y_vec,
                  /*n_rows=*/ctx.modelNSamples_,
                  /*n_samples=*/ctx.nSamples_, /*k=*/n_neighbors);

  cuml4r::pinned_host_vector<float> h_out(d_out.size());
  auto CUML4R_ANONYMOUS_VARIABLE(out_d2h) = cuml4r::async_copy(
    ctx.streamView_.value(), d_out.cbegin(), d_out.cend(), h_out.begin());
  CUDA_RT_CALL(hipStreamSynchronize(ctx.streamView_.value()));

  return Rcpp::NumericVector(h_out.begin(), h_out.end());
}

}  // namespace cuml4r
