#include "async_utils.cuh"
#include "cuda_utils.h"
#include "handle_utils.h"
#include "matrix_utils.h"
#include "pinned_host_vector.h"
#include "preprocessor.h"
#include "stream_allocator.h"

#include <cuml/svm/svm_parameter.h>
#include <thrust/async/copy.h>
#include <thrust/device_vector.h>
#include <cuml/svm/svc.hpp>

#include <Rcpp.h>

#include <memory>
#include <vector>

namespace {

struct ModelCtx {
  using model_t = ML::SVM::SVC<double>;

  // model object must be destroyed first
  std::unique_ptr<raft::handle_t> const handle_;
  std::unique_ptr<model_t> const model_;

  __host__ ModelCtx(std::unique_ptr<raft::handle_t> handle,
                    std::unique_ptr<model_t> model) noexcept
    : handle_(std::move(handle)), model_(std::move(model)) {}
};

}  // namespace

namespace cuml4r {

__host__ SEXP svc_fit(Rcpp::NumericMatrix const& input,
                      Rcpp::NumericVector const& labels, double const cost,
                      int const kernel, double const gamma, double const coef0,
                      int const degree, double const tol, int const max_iter,
                      int const nochange_steps, double const cache_size,
                      Rcpp::NumericVector const& sample_weights,
                      int const verbosity) {
  auto const m = cuml4r::Matrix<>(input, /*transpose=*/true);
  auto const n_samples = m.numCols;
  auto const n_features = m.numRows;

  auto stream_view = cuml4r::stream_allocator::getOrCreateStream();
  auto handle = std::make_unique<raft::handle_t>();
  cuml4r::handle_utils::initializeHandle(*handle, stream_view.value());

  // SVM input
  auto const& h_input = m.values;
  thrust::device_vector<double> d_input(h_input.size());
  auto CUML4R_ANONYMOUS_VARIABLE(input_h2d) = cuml4r::async_copy(
    stream_view.value(), h_input.cbegin(), h_input.cend(), d_input.begin());

  auto h_labels(Rcpp::as<cuml4r::pinned_host_vector<double>>(labels));
  thrust::device_vector<double> d_labels(h_labels.size());
  auto CUML4R_ANONYMOUS_VARIABLE(labels_h2d) = cuml4r::async_copy(
    stream_view.value(), h_labels.cbegin(), h_labels.cend(), d_labels.begin());

  thrust::device_vector<double> d_sample_weights;
  cuml4r::unique_marker sample_weights_h2d;
  if (sample_weights.size() > 0) {
    auto const h_sample_weights(
      Rcpp::as<cuml4r::pinned_host_vector<double>>(sample_weights));
    d_sample_weights.resize(h_sample_weights.size());
    sample_weights_h2d =
      cuml4r::async_copy(stream_view.value(), h_sample_weights.cbegin(),
                         h_sample_weights.cend(), d_sample_weights.begin());
  }

  MLCommon::Matrix::KernelParams kernel_params{
    /*kernel=*/static_cast<MLCommon::Matrix::KernelType>(kernel), degree, gamma,
    coef0};

  // SVM output
  auto svc = std::make_unique<ML::SVM::SVC<double>>(
    *handle, /*C=*/cost, tol, kernel_params, cache_size, max_iter,
    nochange_steps, verbosity);

  svc->fit(d_input.data().get(), /*nrows=*/n_samples, /*ncols=*/n_features,
           d_labels.data().get(),
           d_sample_weights.empty() ? nullptr : d_sample_weights.data().get());

  CUDA_RT_CALL(hipStreamSynchronize(stream_view.value()));

  return Rcpp::XPtr<ModelCtx>(new ModelCtx(std::move(handle), std::move(svc)));
}

__host__ SEXP svc_predict(SEXP model_xptr, Rcpp::NumericMatrix const& input,
                          bool predict_class) {
  auto const m = cuml4r::Matrix<>(input, /*transpose=*/true);
  int const n_samples = m.numCols;
  int const n_features = m.numRows;

  auto ctx = Rcpp::XPtr<ModelCtx>(model_xptr);
  auto const& svc = ctx->model_;
  auto* stream = ctx->handle_->get_stream();

  // input
  auto const& h_input = m.values;
  thrust::device_vector<double> d_input(h_input.size());
  auto CUML4R_ANONYMOUS_VARIABLE(input_h2d) = cuml4r::async_copy(
    stream, h_input.cbegin(), h_input.cend(), d_input.begin());

  // output
  thrust::device_vector<double> d_preds(n_samples);

  if (predict_class) {
    svc->predict(/*input=*/d_input.data().get(), /*n_rows=*/n_samples,
                 /*c_cols=*/n_features, /*preds=*/d_preds.data().get());
  } else {
    ML::SVM::svcPredict(
      /*handle=*/*ctx->handle_, /*input=*/d_input.data().get(),
      /*n_rows=*/n_samples,
      /*c_cols=*/n_features, /*kernel_parames=*/svc->kernel_params,
      /*model=*/svc->model, /*preds=*/d_preds.data().get(),
      /*buffer_size=*/svc->param.cache_size, /*predict_class=*/false);
  }

  cuml4r::pinned_host_vector<double> h_preds(n_samples);
  auto CUML4R_ANONYMOUS_VARIABLE(preds_d2h) = cuml4r::async_copy(
    stream, d_preds.cbegin(), d_preds.cend(), h_preds.begin());
  CUDA_RT_CALL(hipStreamSynchronize(stream));

  return Rcpp::NumericVector(h_preds.begin(), h_preds.end());
}

}  // namespace cuml4r
